#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result

__global__ void upsweep_kernel(int* result, int two_d, int two_dplus1, int N) {
    int my_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = my_idx * two_dplus1;
    //TODO: create fast shared memory?
    if (i < N) {
        result[i + two_dplus1 - 1] += result[i + two_d - 1];
    }

}

__global__ void downsweep_kernel(int* result, int two_d, int two_dplus1, int N) {
    int my_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = my_idx * two_dplus1;
    if (i < N) {
        int t = result[i + two_d - 1];
        result[i + two_d - 1] = result[i + two_dplus1 - 1];
        result[i + two_dplus1 -1 ] += t;
    }

}

__global__ void set_last_zero_kernel(int* result, int N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {

        result[N - 1] = 0;
    }
}

void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    N = nextPow2(N);
    int block_size = 32;

    // upsweep phase
    for (int two_d = 1;  two_d <= N / 2; two_d *= 2) {
        int two_dplus1 = 2 * two_d;
        
        int num_index = (N + 1) / two_dplus1;
        int grid_size = (num_index + block_size - 1) / block_size;
        
        upsweep_kernel<<<grid_size, block_size>>>(result, two_d, two_dplus1, N);
        
        hipDeviceSynchronize();
        hipError_t kernel_return_code = hipGetLastError();
        if (kernel_return_code != hipSuccess) {
            printf("%s\n", hipGetErrorString(kernel_return_code));
        }
    }

    // hipMemcpy(result, device_result, N * sizeof(int), hipMemcpyDeviceToHost);
    
    set_last_zero_kernel<<<1, 1>>>(result, N);
    hipDeviceSynchronize();

    // hipMemcpy(device_result, result, N * sizeof(int), hipMemcpyHostToDevice);

    // downsweep phase
    for (int two_d = N/2; two_d >= 1; two_d /= 2) {
        int two_dplus1 = 2 * two_d;
        
        int num_index = (N + 1)  / two_dplus1;
        int grid_size = (num_index + block_size - 1) / block_size;

        downsweep_kernel<<<grid_size, block_size>>>(result, two_d, two_dplus1, N);
        hipDeviceSynchronize();
        hipError_t kernel_return_code = hipGetLastError();
        if (kernel_return_code != hipSuccess) {
            printf("%s\n", hipGetErrorString(kernel_return_code));
        }

    }
    // hipMemcpy(result, device_result, N * sizeof(int), hipMemcpyDeviceToHost);
    // hipFree(device_result);


}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

__global__ void next_equal_kernel(int* input, int N, int* output, int* output_copy) {
    int my_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (my_idx < N - 1) {
        output[my_idx] = (input[my_idx] == input[my_idx + 1]);
        output_copy[my_idx] = (input[my_idx] == input[my_idx + 1]);
    }
    else {
        output[my_idx] = 0;
        output_copy[my_idx] = 0;
    }
}

__global__ void get_repeat_index_kernel(int* next_equal, int* next_equal_prefix_s, int* output) {
    int my_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (next_equal[my_idx] == 1) {
        output[next_equal_prefix_s[my_idx]] = my_idx;
    }

}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    int rounded_length = nextPow2(length);
    int* device_next_equal;
    int* device_next_equal_prefix_s;
    hipMalloc(&device_next_equal, sizeof(int) * rounded_length);
    hipMalloc(&device_next_equal_preefix_s, sizeof(int) * rounded_length);

    next_equal_kernel<<<>>>(device_input, length, device_next_equal, device_next_equal_prefix_s);
    hipDeviceSynchronize();

    exclusive_scan(device_next_equal, length, device_next_equal_prefix_s);

    get_repeat_index_kernel<<<,>>>(device_next_equal, device_next_equal_prefix_s, device_output);
    hipDeviceSynchronize();

    hipFree(device_next_equal);

    int output_length;
    hipMemcpy(&output_length, device_next_equal_prefix_s + length - 1, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_next_equal_prefix_s);

    return output_length; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
